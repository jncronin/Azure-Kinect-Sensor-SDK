
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <float.h>
#include <stdlib.h>
#include <time.h>

#define E_PI 3.141592f

#define PROFILE 0

__device__ static void GetPhase(const float* d, float* phase, float* amplitude, float* offset)
{
    // See https://math.stackexchange.com/questions/118526/fitting-a-sine-wave-of-known-frequency-through-three-points
    float c = (d[0] + d[2]) / 2.0f;
    *offset = c;
    float a = sqrtf((d[0] - c) * (d[0] - c) + (d[1] - c) * (d[1] - c));
    *amplitude = a;
    float b = atan2f(d[0] - c, d[1] - c);
    *phase = b;
}


__device__ float GetNFOVData(int x, int y, int frame, const unsigned char* image)
{
    const int frame_width = 640;
    const int frame_height = 576;
    const int frame_stride = frame_width * 8 / 5;
    const int metadata_length = 256;
    int offset = (frame_height * frame_stride + metadata_length) * frame + metadata_length +
        y * frame_stride;
    int block_of_8 = x / 5;
    int idx = offset + block_of_8 * 8 + x % 5;

    int d = (int)image[idx];
    if (d >= 64)
        d = 64 - d;
    return d;
}

__device__ float GetWFOVBinnedData(int x, int y, int frame, const unsigned char* image)
{
    int x_offset = (160 * (frame + 1)) % 512;
    int src_x = x + x_offset;
    if (src_x >= 512) src_x -= 512;

    int pix_id = src_x + y * 512 + frame * 512 * 512;
    int block_of_8 = pix_id / 5;
    int block_id = pix_id % 5;
    int src_id = block_of_8 * 8 + block_id;

    int dval = (int)image[src_id];

    // proc
    if (dval >= 64) dval = 64 - dval;

    return dval;
}

__device__ static inline float GetNFOVDistance(const float* phases, float* err)
{
    /* Calibration gives us:
        d1 = 0.734 * phase1 - 0.300 
        d2 = 0.778 * phase2 - 0.150
        d3 = 2.866 * phase3 - 1.053
        
        For max dist of 3.86m (as per data sheet), we get
        max phase1 = 5.66 * 2pi
        max phase2 = 5.42 * 2pi
        max phase3 = 1.71 * 2pi */

    const int f1n = 5;
    const int f2n = 5;
    const int f3n = 1;

    float best_err = FLT_MAX;
    float best_dist = 0.0f;

    // brute force algorithm as per https://medium.com/chronoptics-time-of-flight/phase-wrapping-and-its-solution-in-time-of-flight-depth-sensing-493aa8b21c42
    for (int k = 0; k <= f3n; k++)
    {
        for (int j = 0; j <= f2n; j++)
        {
            for (int i = 0; i <= f1n; i++)
            {
                float d1 = 0.734f / 2.0f / E_PI * (phases[0] + (float)i * 2.0f * E_PI) - 0.300f;
                float d2 = 0.778f / 2.0f / E_PI * (phases[1] + (float)j * 2.0f * E_PI) - 0.357f;
                float d3 = 2.866f / 2.0f / E_PI * (phases[2] + (float)k * 2.0f * E_PI) - 1.053f;

                float d_mean = (d1 + d2 + d3) / 3.0f;
                float d_var = ((d1 - d_mean) * (d1 - d_mean) + (d2 - d_mean) * (d2 - d_mean) + (d3 - d_mean) * (d3 - d_mean)) / 3.0f;
                // TODO: profile to see which of these is best
#if 0
                if (d_var < best_err)
                {
                    best_err = d_var;
                    best_dist = d_mean;
#                }
#endif

#if 1
                best_dist = d_var < best_err ? d_mean : best_dist;
                best_err = d_var < best_err ? d_var : best_err;
#endif
            }
        }
    }

    *err = best_err;

    return best_dist;
}

__device__ static inline float GetWFOVBinnedDistance(const float* phases, float* err, float x)
{
    /* Calibration gives us (d in mm):
        d1 = 781.6 * phase1 + 179.73 - 4.528 * x 
        d2 = 817.8 * phase2 + 68.11 - 4.204 * x
        d3 = 3124.2 * phase3 + 538.74 - 10.39 * x
        
        For max dist of 3.86m (as per data sheet), we get
        max phase1 = 5.66 * 2pi
        max phase2 = 5.42 * 2pi
        max phase3 = 1.71 * 2pi */

    const int f1n = 5;
    const int f2n = 5;
    const int f3n = 2;

    float best_err = FLT_MAX;
    float best_dist = 0.0f;

    // brute force algorithm as per https://medium.com/chronoptics-time-of-flight/phase-wrapping-and-its-solution-in-time-of-flight-depth-sensing-493aa8b21c42
    for (int k = 0; k <= f3n; k++)
    {
        for (int j = 0; j <= f2n; j++)
        {
            for (int i = 0; i <= f1n; i++)
            {
                float d1 = 779.0995f / 2.0f / E_PI * (phases[0] + (float)i * 2.0f * E_PI) - 74.3193f - 3.3783f * x;
                float d2 = 824.3284f / 2.0f / E_PI * (phases[1] + (float)j * 2.0f * E_PI) - 93.5419f - 3.5326f * x;
                float d3 = 2702.1864f / 2.0f / E_PI * (phases[2] + (float)k * 2.0f * E_PI) - 816.6109f - 3.1806f * x;

                d1 /= 1000.0f;
                d2 /= 1000.0f;
                d3 /= 1000.0f;

                float d_mean = (d1 + d2 + d3) / 3.0f;
                float d_var = ((d1 - d_mean) * (d1 - d_mean) + (d2 - d_mean) * (d2 - d_mean) + (d3 - d_mean) * (d3 - d_mean)) / 3.0f;
                // TODO: profile to see which of these is best
#if 0
                if (d_var < best_err)
                {
                    best_err = d_var;
                    best_dist = d_mean;
                }
#endif

#if 1
                best_dist = d_var < best_err ? d_mean : best_dist;
                best_err = d_var < best_err ? d_var : best_err;
#endif
            }
        }
    }

    *err = best_err;

    return best_dist;
}


#if PROFILE
#define PROFILE_START(a) unsigned int pstart ## a, pend ## a; pstart ## a = clock();
#define PROFILE_END(a) pend ## a = clock(); dev_times ## a ## [outidx] = pend ## a - pstart ## a;
#else
#define PROFILE_START(a)
#define PROFILE_END(a)
#endif

// buffer sizes
const int NFOVUnbinned_in_count = 1024 * 576 * 9;
const int NFOVUnbinned_out_count = 640 * 576;

const int WFOVBinned_in_count = 3777232;
const int WFOVBinned_out_count = 512 * 512;


__global__ void NFOVUnbinnedKernel(unsigned short int* depth_out,
    unsigned short int* ir_out,
    const unsigned char* data,
    int xbin, int ybin
#if PROFILE    
    , unsigned int *dev_times1, unsigned int *dev_times2, unsigned int *dev_times3
#endif
    )
{
    int outidx = threadIdx.x + blockIdx.x * blockDim.x;

    const int frame_width = 640;
    const int frame_height = 576;

    int x = (outidx % (frame_width / xbin)) * xbin;
    int y = (outidx / (frame_width / xbin)) * ybin;

    outidx = x + y * frame_width;

    float phases[3];
    float offsets[3];
    float amplitudes[3];
    float d[9];

    PROFILE_START(1);
    PROFILE_START(2);
    for (int i = 0; i < 9; i++)
    {
        d[i] = GetNFOVData(x, y, i, data);
    }

    for (int i = 0; i < 3; i++)
    {
        GetPhase(&d[i * 3], &phases[i], &amplitudes[i], &offsets[i]);
    }
    PROFILE_END(2);

    // Apply a fiddle factor based upon experimentation to account for time delay
    //  between imaging each column of the IR image
    phases[0] = fmodf(phases[0] - 2.7f * (float)x / 200.0f, E_PI * 2.0f);
    phases[1] = fmodf(phases[1] - 2.55f * (float)x / 200.0f, E_PI * 2.0f);
    phases[2] = fmodf(phases[2] - 1.05f * (float)x / 200.0f, E_PI * 2.0f);
    if (phases[0] < 0.0f) phases[0] += E_PI * 2.0f;
    if (phases[1] < 0.0f) phases[1] += E_PI * 2.0f;
    if (phases[2] < 0.0f) phases[2] += E_PI * 2.0f;

    PROFILE_START(3);
    float err;
    float dist = GetNFOVDistance(phases, &err);
    PROFILE_END(3);
    float irf = fabsf((offsets[0] + offsets[1] + offsets[2]) / 3.0f / dist / dist * 1000.0f);

    unsigned short int depth_val = (unsigned short int)(dist * 1000.0f); // mm distance
    unsigned short int ir_val = (unsigned short int)irf;

    // Masking calculations

    // First, NFOV uses circular lens mask
    int mask_lens = (((float)x / (float)(frame_width / 2) - 1.0f) * ((float)x / (float)(frame_width / 2) - 1.0f) +
        ((float)y / (float)(frame_height / 2) - 1.0f) * ((float)y / (float)(frame_height / 2) - 1.0f)) < 1.0f ? 1 : 0;

    // Then variance in the output values
    int mask_err = err < 0.05f ? 1 : 0;

    // Finally, assume that the amplitude of the returned signal is inversely proportional to the square of
    //  the distance
    int mask_amp = ((amplitudes[0] + amplitudes[1] + amplitudes[2]) * dist * dist) < 200.0f ? 1 : 0;

    unsigned short mask = (unsigned short)(mask_lens * mask_err * mask_amp);

    depth_val *= mask;
    ir_val *= mask_lens;

    for(int j = 0; j < ybin; j++)
    {
        for(int i = 0; i < xbin; i++)
        {
            depth_out[outidx + i + j * frame_width] = depth_val;
            ir_out[outidx + i + j * frame_width] = ir_val;
        }
    }

    PROFILE_END(1);
}

__global__ void WFOVBinnedKernel(unsigned short int* depth_out,
    unsigned short int* ir_out,
    const unsigned char* data,
    int xbin, int ybin
#if PROFILE    
    , unsigned int *dev_times1, unsigned int *dev_times2, unsigned int *dev_times3
#endif
    )
{
    int outidx = threadIdx.x + blockIdx.x * blockDim.x;

    const int frame_width = 512;
    const int frame_height = 512;

    int x = (outidx % (frame_width / xbin)) * xbin;
    int y = (outidx / (frame_width / xbin)) * ybin;

    outidx = x + y * frame_width;

    float phases[3];
    float offsets[3];
    float amplitudes[3];
    float d[9];

    PROFILE_START(1);
    PROFILE_START(2);
    for (int i = 0; i < 9; i++)
    {
        d[i] = GetWFOVBinnedData(x, y, i, data);
    }

    for (int i = 0; i < 3; i++)
    {
        GetPhase(&d[i * 3], &phases[i], &amplitudes[i], &offsets[i]);
    }
    PROFILE_END(2);

    phases[0] = fmodf(phases[0], E_PI * 2.0f);
    phases[1] = fmodf(phases[1], E_PI * 2.0f);
    phases[2] = fmodf(phases[2], E_PI * 2.0f);
    if (phases[0] < 0.0f) phases[0] += E_PI * 2.0f;
    if (phases[1] < 0.0f) phases[1] += E_PI * 2.0f;
    if (phases[2] < 0.0f) phases[2] += E_PI * 2.0f;

    PROFILE_START(3);
    float err;
    float dist = GetWFOVBinnedDistance(phases, &err, x);
    PROFILE_END(3);
    float irf = fabsf((offsets[0] + offsets[1] + offsets[2]) / 3.0f / dist / dist * 1000.0f);

    unsigned short int depth_val = (unsigned short int)(dist * 1000.0f); // mm distance
    unsigned short int ir_val = (unsigned short int)irf;

    // Masking calculations

    // First, NFOV uses circular lens mask
    int mask_lens = (((float)x / (float)(frame_width / 2) - 1.0f) * ((float)x / (float)(frame_width / 2) - 1.0f) +
        ((float)y / (float)(frame_height / 2) - 1.0f) * ((float)y / (float)(frame_height / 2) - 1.0f)) < 1.0f ? 1 : 0;

    // Then variance in the output values
    int mask_err = err < 0.01f ? 1 : 0;

    // Finally, assume that the amplitude of the returned signal is inversely proportional to the square of
    //  the distance
    float amp_dist = (amplitudes[0] + amplitudes[1] + amplitudes[2]) * dist * dist;
    int mask_amp = (amp_dist >= 10.0f && amp_dist < 50.0f) ? 1 : 0;

    unsigned short mask = (unsigned short)(mask_lens * mask_err * mask_amp);

    depth_val *= mask;
    ir_val *= mask_lens;

    for(int j = 0; j < ybin; j++)
    {
        for(int i = 0; i < xbin; i++)
        {
            depth_out[outidx + i + j * frame_width] = depth_val;
            ir_out[outidx + i + j * frame_width] = ir_val;
        }
    }

    PROFILE_END(1);
}


// buffers to hold device data
unsigned char* dev_data;
unsigned short* dev_ir_out;
unsigned short* dev_depth_out;

unsigned int* dev_times1;
unsigned int* dev_times2;
unsigned int* dev_times3;

const int nthreads = 128;

extern "C" {

// Function to call the kernel
void RunNFOVUnbinnedCalculation(unsigned short int* depth_out,
    unsigned short int* ir_out,
    const unsigned char* data,
    int xbin, int ybin)
{
    // TODO: add error checking here
    hipError_t cudaStatus = hipMemcpy(dev_data, data, NFOVUnbinned_in_count * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return;
    }

    NFOVUnbinnedKernel <<<NFOVUnbinned_out_count / nthreads / xbin / ybin, nthreads>>> (dev_depth_out, dev_ir_out, dev_data,
        xbin, ybin
#if PROFILE
        , dev_times1, dev_times2, dev_times3
#endif
        );

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }

    if(depth_out)
    {
        cudaStatus = hipMemcpy(depth_out, dev_depth_out, NFOVUnbinned_out_count * sizeof(unsigned short), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return;
        }
    }

    if(ir_out)
    {
        cudaStatus = hipMemcpy(ir_out, dev_ir_out, NFOVUnbinned_out_count * sizeof(unsigned short), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return;
        }
    }

#if PROFILE
    unsigned int* times1 = (unsigned int*)malloc(NFOVUnbinned_out_count * sizeof(unsigned int));
    unsigned int* times2 = (unsigned int*)malloc(NFOVUnbinned_out_count * sizeof(unsigned int));
    unsigned int* times3 = (unsigned int*)malloc(NFOVUnbinned_out_count * sizeof(unsigned int));
    cudaMemcpy(times1, dev_times1, NFOVUnbinned_out_count * sizeof(unsigned int), cudaMemcpyDeviceToHost);
    cudaMemcpy(times2, dev_times2, NFOVUnbinned_out_count * sizeof(unsigned int), cudaMemcpyDeviceToHost);
    cudaMemcpy(times3, dev_times3, NFOVUnbinned_out_count * sizeof(unsigned int), cudaMemcpyDeviceToHost);

    unsigned int times1_worst = 0;
    unsigned int times2_worst = 0;
    unsigned int times3_worst = 0;

    for (int i = 0; i < NFOVUnbinned_out_count; i++)
    {
        if (times1[i] >= times1_worst) times1_worst = times1[i];
        if (times2[i] >= times2_worst) times2_worst = times2[i];
        if (times3[i] >= times3_worst) times3_worst = times3[i];
    }

    printf("t1: %i, t2: %i, t3: %i\n", times1_worst, times2_worst, times3_worst);

    free(times1);
    free(times2);
    free(times3);
#endif
}

void RunWFOVBinnedCalculation(unsigned short int* depth_out,
    unsigned short int* ir_out,
    const unsigned char* data,
    int xbin, int ybin)
{
    // TODO: add error checking here
    hipError_t cudaStatus = hipMemcpy(dev_data, data, WFOVBinned_in_count * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return;
    }

    WFOVBinnedKernel <<<WFOVBinned_out_count / nthreads / xbin / ybin, nthreads>>> (dev_depth_out, dev_ir_out, dev_data,
        xbin, ybin
#if PROFILE
        , dev_times1, dev_times2, dev_times3
#endif
        );

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }

    if(depth_out)
    {
        cudaStatus = hipMemcpy(depth_out, dev_depth_out, WFOVBinned_out_count * sizeof(unsigned short), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return;
        }
    }

    if(ir_out)
    {
        cudaStatus = hipMemcpy(ir_out, dev_ir_out, WFOVBinned_out_count * sizeof(unsigned short), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            return;
        }
    }

#if PROFILE
    unsigned int* times1 = (unsigned int*)malloc(WFOVBinned_out_count * sizeof(unsigned int));
    unsigned int* times2 = (unsigned int*)malloc(WFOVBinned_out_count * sizeof(unsigned int));
    unsigned int* times3 = (unsigned int*)malloc(WFOVBinned_out_count * sizeof(unsigned int));
    cudaMemcpy(times1, dev_times1, WFOVBinned_out_count * sizeof(unsigned int), cudaMemcpyDeviceToHost);
    cudaMemcpy(times2, dev_times2, WFOVBinned_out_count * sizeof(unsigned int), cudaMemcpyDeviceToHost);
    cudaMemcpy(times3, dev_times3, WFOVBinned_out_count * sizeof(unsigned int), cudaMemcpyDeviceToHost);

    unsigned int times1_worst = 0;
    unsigned int times2_worst = 0;
    unsigned int times3_worst = 0;

    for (int i = 0; i < WFOVBinned_out_count; i++)
    {
        if (times1[i] >= times1_worst) times1_worst = times1[i];
        if (times2[i] >= times2_worst) times2_worst = times2[i];
        if (times3[i] >= times3_worst) times3_worst = times3[i];
    }

    printf("t1: %i, t2: %i, t3: %i\n", times1_worst, times2_worst, times3_worst);

    free(times1);
    free(times2);
    free(times3);
#endif
}


// Init function
void InitNFOVUnbinnedCalculation()
{
    hipSetDevice(0);
    hipMalloc(&dev_data, NFOVUnbinned_in_count * sizeof(unsigned char));
    hipMalloc(&dev_ir_out, NFOVUnbinned_out_count * sizeof(unsigned short int));
    hipMalloc(&dev_depth_out, NFOVUnbinned_out_count * sizeof(unsigned short int));

#ifdef PROFILE
    hipMalloc(&dev_times1, NFOVUnbinned_out_count * sizeof(unsigned int));
    hipMalloc(&dev_times2, NFOVUnbinned_out_count * sizeof(unsigned int));
    hipMalloc(&dev_times3, NFOVUnbinned_out_count * sizeof(unsigned int));
#endif
}

void InitWFOVBinnedCalculation()
{
    hipSetDevice(0);
    hipMalloc(&dev_data, WFOVBinned_in_count * sizeof(unsigned char));
    hipMalloc(&dev_ir_out, WFOVBinned_out_count * sizeof(unsigned short int));
    hipMalloc(&dev_depth_out, WFOVBinned_out_count * sizeof(unsigned short int));

#ifdef PROFILE
    hipMalloc(&dev_times1, WFOVBinned_out_count * sizeof(unsigned int));
    hipMalloc(&dev_times2, WFOVBinned_out_count * sizeof(unsigned int));
    hipMalloc(&dev_times3, WFOVBinned_out_count * sizeof(unsigned int));
#endif
}


// Dealloc function
void DeinitNFOVUnbinnedCalculation()
{
    if (dev_data)
    {
        hipFree(dev_data);
        dev_data = NULL;
    }
    if (dev_ir_out)
    {
        hipFree(dev_ir_out);
        dev_ir_out = NULL;
    }
    if (dev_depth_out)
    {
        hipFree(dev_depth_out);
        dev_depth_out = NULL;
    }

#if PROFILE
    if (dev_times1)
    {
        cudaFree(dev_times1);
        dev_times1 = NULL;
    }
    if (dev_times2)
    {
        cudaFree(dev_times2);
        dev_times2 = NULL;
    }
    if (dev_times3)
    {
        cudaFree(dev_times3);
        dev_times3 = NULL;
    }
#endif

}

void DeinitWFOVBinnedCalculation()
{
    if (dev_data)
    {
        hipFree(dev_data);
        dev_data = NULL;
    }
    if (dev_ir_out)
    {
        hipFree(dev_ir_out);
        dev_ir_out = NULL;
    }
    if (dev_depth_out)
    {
        hipFree(dev_depth_out);
        dev_depth_out = NULL;
    }

#if PROFILE
    if (dev_times1)
    {
        cudaFree(dev_times1);
        dev_times1 = NULL;
    }
    if (dev_times2)
    {
        cudaFree(dev_times2);
        dev_times2 = NULL;
    }
    if (dev_times3)
    {
        cudaFree(dev_times3);
        dev_times3 = NULL;
    }
#endif

}


}
